#include "hip/hip_runtime.h"
/*
 * inference-101
 */

#include "cudaRGB.h"

//-------------------------------------------------------------------------------------------------------------------------

__global__ void RGBToRGBAf(uint8_t* srcImage,
                           float4* dstImage,
                           uint32_t width,       uint32_t height)
{
    int x, y, pixel;

    x = (blockIdx.x * blockDim.x) + threadIdx.x;
    y = (blockIdx.y * blockDim.y) + threadIdx.y;
    pixel = y * width + x;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);

	const float s = 1;
	dstImage[pixel]     = make_float4(srcImage[pixel*3] * s, srcImage[ pixel*3 + 1] * s, srcImage[ pixel*3 + 2] * s, 0.0f);
}

hipError_t cudaRGBToRGBAf( uint8_t* srcDev, float4* destDev, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	const dim3 blockDim(128,1,1);
	const dim3 gridDim(width/blockDim.x, height/blockDim.y, 1);

	RGBToRGBAf<<<gridDim, blockDim>>>( (uint8_t*)srcDev, destDev, width, height );

	return CUDA(hipGetLastError());
}


//-------------------------------------------------------------------------------------------------------------------------

__global__ void BAYER_GR8toRGBA(uint8_t* srcImage,
                           float4* dstImage,
                           uint32_t width,       uint32_t height)
{
    int x, y, pixel;
	bool lineOdd, pixelOdd;

    x = (blockIdx.x * blockDim.x) + threadIdx.x;
    y = (blockIdx.y * blockDim.y) + threadIdx.y;
    pixel = y * width + x;
    
    pixelOdd = (pixel % 2) ? true : false;
    lineOdd = ((pixel / width) % 2) ? true : false;
    
    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);

#if 1
	// Convert to RGB
	if ((lineOdd) && (!pixelOdd))        
		dstImage[pixel] = make_float4(srcImage[pixel+width], srcImage[pixel], srcImage[pixel-1], 0.0f); // Green Info
	else if ((lineOdd) && (pixelOdd))   
		dstImage[pixel] = make_float4(srcImage[pixel+width+1], srcImage[pixel+1], srcImage[pixel], 0.0f); // Blue Info
#if 1
	if ((!lineOdd) && (!pixelOdd)) 
		dstImage[pixel] = make_float4(srcImage[pixel], srcImage[pixel-1], srcImage[pixel+width+1], 0.0f); // Red Info
	else if ((!lineOdd) && (pixelOdd)) 
		dstImage[pixel] = make_float4(srcImage[pixel+1], srcImage[pixel], srcImage[pixel+width], 0.0f); // Green Info
#endif

#else
	// Monochrome output
	dstImage[pixel]     = make_float4(srcImage[pixel], srcImage[ pixel], srcImage[ pixel], 0.0f);
#endif
}

hipError_t cudaBAYER_GR8toRGBA( uint8_t* srcDev, float4* destDev, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	const dim3 blockDim(128,1,1);
	const dim3 gridDim(width/blockDim.x, height/blockDim.y, 1);

	BAYER_GR8toRGBA<<<gridDim, blockDim>>>( (uint8_t*)srcDev, destDev, width, height );

	return CUDA(hipGetLastError());
}


